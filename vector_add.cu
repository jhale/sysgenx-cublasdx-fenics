#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
        C[i] = A[i] + B[i];
}

int main(void) {
    // Number of elements in the vectors
    int numElements = 50000;
    size_t size = numElements * sizeof(float);

    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate device memory
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy input vectors from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch the CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    float maxError = 0.0f;
    for (int i = 0; i < numElements; i++) {
        maxError = fmax(maxError, fabs(h_C[i] - (h_A[i] + h_B[i])));
    }
    printf("Max error: %f\n", maxError);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Test PASSED\n");
    return 0;
} 