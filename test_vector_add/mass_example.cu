#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <vector>

#include <basix/finite-element.h>
#include <basix/quadrature.h>

#include <dolfinx.h>
#include <dolfinx/fem/Function.h>

#include <cublasdx.hpp>
#include <hip/hip_runtime_api.h>

using T = double;

constexpr int P = 4;
constexpr std::size_t num_dofs = (P + 1) * (P + 2) * (P + 3) / 6;
constexpr std::size_t batch_size = 64;
constexpr std::size_t m = num_dofs;
constexpr std::size_t n = batch_size;
constexpr std::size_t k = num_dofs;
constexpr std::size_t num_quadrature_points = num_dofs;

constexpr std::size_t num_elements = 32 * 32 * 32 * 6;

template <class GEMM, class GEMM_T>
__global__ void gemm_kernel_shared(const T* phi, const T* u, const T* c,
                                   const T* detJ, T* output, size_t num_dofs)
{
  extern __shared__ __align__(16) char smem[];

  // Get block index
  const int block_idx = blockIdx.x;
  const int thread_idx = threadIdx.x;

  const size_t u_offset = block_idx * batch_size * num_dofs;
  const size_t c_offset = block_idx * batch_size * num_dofs;
  const size_t detJ_offset = block_idx * batch_size * num_quadrature_points;

  //  Copy tensors to shared memory
  auto phi_tensor = cublasdx::make_tensor(phi, GEMM::get_layout_gmem_a());
  auto u_tensor
      = cublasdx::make_tensor(u + u_offset, GEMM::get_layout_gmem_b());
  auto c_tensor
      = cublasdx::make_tensor(c + c_offset, GEMM::get_layout_gmem_c());

  auto [smem_a, smem_b, smem_c] = cublasdx::slice_shared_memory<GEMM>(smem);
  auto a_shared_tensor
      = cublasdx::make_tensor(smem_a, GEMM::get_layout_smem_a());
  auto b_shared_tensor
      = cublasdx::make_tensor(smem_b, GEMM::get_layout_smem_b());
  auto c_shared_tensor
      = cublasdx::make_tensor(smem_c, GEMM::get_layout_smem_c());

  using alignment = cublasdx::alignment_of<GEMM>;

  // First GEMM: phi * U
  cublasdx::copy<GEMM, alignment::a>(phi_tensor, a_shared_tensor);
  cublasdx::copy<GEMM, alignment::b>(u_tensor, b_shared_tensor);
  cublasdx::copy_wait();

  T alpha1 = 1.0;
  T beta1 = 0.0;
  GEMM().execute(alpha1, a_shared_tensor, b_shared_tensor, beta1,
                 c_shared_tensor);

  //  Scale by Jacobian determinant
  for (int i = thread_idx; i < num_quadrature_points * batch_size;
       i += blockDim.x)
  {
    c_shared_tensor[i] = c_shared_tensor[i] * detJ[i];
  }

  __syncthreads();

  T alpha2 = 1.0;
  T beta2 = 0.0;
  GEMM_T().execute(alpha2, a_shared_tensor, c_shared_tensor, beta2,
                   b_shared_tensor);

  // Copy result back to global memory
  auto out_global_tensor
      = cublasdx::make_tensor(output + c_offset, GEMM::get_layout_gmem_c());
  cublasdx::copy<GEMM, alignment::c>(b_shared_tensor, out_global_tensor);
}

int main(int argc, char* argv[])
{
  {
    MPI_Init(&argc, &argv);
    [[maybe_unused]] constexpr int Arch = 700;

    dolfinx::init_logging(argc, argv);
    auto part = mesh::create_cell_partitioner(mesh::GhostMode::shared_facet);
    auto mesh = std::make_shared<mesh::Mesh<T>>(mesh::create_box<T>(
        MPI_COMM_WORLD, {{{0.0, 0.0, 0.0}, {1.0, 1.0, 1.0}}}, {32, 32, 32},
        mesh::CellType::hexahedron, part));

    // Tabulation of basis functions
    basix::FiniteElement element = basix::create_element<T>(
        basix::element::family::P, basix::cell::type::tetrahedron, P,
        basix::element::lagrange_variant::equispaced,
        basix::element::dpc_variant::unset, false);

    auto [x, weights] = basix::quadrature::make_quadrature<T>(
        basix::quadrature::type::Default, basix::cell::type::tetrahedron,
        basix::polyset::type::standard, 2 * P);

    auto [table, shape] = element.tabulate(1, points, {weights.size(), 3});

    auto V
        = std::make_shared<fem::FunctionSpace<T>>(fem::create_functionspace<T>(
            mesh, std::make_shared<fem::FiniteElement<T>>(element)));

    auto u_function = std::make_shared<fem::Function<T>>(V);
    u_function->interpolate(
        [](auto x) -> std::pair<std::vector<T>, std::vector<std::size_t>>
        {
          std::vector<T> f;
          for (std::size_t p = 0; p < x.extent(1); ++p)
          {
            f.push_back(x(0, p) + x(1, p) + x(2, p));
          }

          return {f, {f.size()}};
        });

    auto arrangement
        = cublasdx::Arrangement<cublasdx::row_major, cublasdx::row_major,
                                cublasdx::row_major>();
    auto size = cublasdx::Size<m, n, k>();
    auto precision = cublasdx::Precision<T>();
    auto type = cublasdx::Type<cublasdx::type::real>();
    auto function = cublasdx::Function<cublasdx::function::MM>();
    auto sm = cublasdx::SM<Arch>();
    auto block = cublasdx::Block();
    auto block_dim = cublasdx::BlockDim<256>();

    using GEMM = decltype(size + precision + type + arrangement + function + sm
                          + block + block_dim);

    auto transpose = cublasdx::Arrangement<cublasdx::arrangement::col_major,
                                           cublasdx::arrangement::row_major,
                                           cublasdx::arrangement::col_major>();
    using GEMM_T = decltype(size + precision + type + transpose + function + sm
                            + block + block_dim);

    // Allocate memory
    constexpr auto num_quadrature_points = num_dofs;
    constexpr auto phi_size
        = num_elements * num_quadrature_points; // Batch size
    constexpr auto u_size = num_elements * num_dofs;
    constexpr auto global_c_size = num_elements * num_dofs;
    constexpr auto detJ_size = num_elements * num_quadrature_points;

    T* phi;  // basis functions
    T* u;    // coefficients DG style
    T* detJ; // Jacobian determinant
    T* c;    // result

    hipMallocManaged(&phi, phi_size * sizeof(T));
    hipMallocManaged(&u, u_size * sizeof(T));
    hipMallocManaged(&c, global_c_size * sizeof(T));
    hipMallocManaged(&detJ, detJ_size * sizeof(T));

    // Initialize phi with basis functions
    for (int i = 0; i < phi_size; ++i)
    {
      phi[i] = table[i];
    }

    // Initialize detJ with Jacobian determinant (Assumed to be 1 for now)
    for (int i = 0; i < detJ_size; ++i)
    {
      detJ[i] = 1.0 / num_elements; // TODO: compute
    }

    // Copy u coefficients to device
    std::shared_ptr dofmap = V->dofmap();
    auto [dof_indices, unrolled] = dofmap->dof_indices();

    for (int i = 0; i < u_size; ++i)
    {
      u[i] = u_function->vector()->mutable_array()[dof_indices[i]];
    }

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Calculate grid and block dimensions
    constexpr int block_dim = 256; // Use the GEMM block size
    constexpr int num_batches = (num_elements + batch_size - 1) / batch_size;
    constexpr int grid_size = num_batches;

    // Record start event
    hipEventRecord(start);

    // Launch kernel

    for (int i = 0; i < 10; ++i)
    {
      gemm_kernel_shared<GEMM, GEMM_T>
          <<<grid_size, block_dim, cublasdx::get_shared_storage_size<GEMM>()>>>(
              phi, u, c, detJ, c, u_size);
      hipDeviceSynchronize();
    }

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
      std::cerr << "Kernel launch failed: " << hipGetErrorString(err)
                << std::endl;
      return 1;
    }

    // Synchronize to ensure kernel completes
    hipDeviceSynchronize();

    // Record stop event
    hipEventRecord(stop);

    // Synchronize to ensure timing is accurate
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print timing information
    std::cout << "Kernel execution time: " << milliseconds << " ms"
              << std::endl;
    std::cout << "Throughput: "
              << 10 * (3 * global_c_size * sizeof(T) / (milliseconds / 1000.0))
                     / 1e9
              << " GB/s" << std::endl;
    std::cout << "Grid size: " << grid_size << ", Block size: " << block_dim
              << std::endl;
    std::cout << "Number of batches: " << num_batches << std::endl;

    // Clean up CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Clean up memory
    hipFree(phi);
    hipFree(u);
    hipFree(c);
    hipFree(detJ);
  }

  MPI_Finalize();
  return 0;
}
